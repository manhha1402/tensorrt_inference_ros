#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cfloat>
#include <map>
#include "tensorrt_infer_core/similarity.hpp"
namespace tensorrt_infer_core
{
    // Custom atomicMax for doubles (since CUDA does not support atomicMax for doubles by default)
    __device__ double atomicMax_double(double *address, double val)
    {
        unsigned long long int *address_as_ull = (unsigned long long int *)address;
        unsigned long long int old = *address_as_ull, assumed;

        do
        {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed, __double_as_longlong(max(val, __longlong_as_double(assumed))));
        } while (assumed != old);

        return __longlong_as_double(old);
    }
    __global__ void compute_dot_products(const double *vectors, const double *target_vector, double *dot_products, int num_vectors, int vector_size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < num_vectors)
        {
            double dot_product = 0.0;
            for (int i = 0; i < vector_size; ++i)
            {
                dot_product += vectors[idx * vector_size + i] * target_vector[i];
            }
            dot_products[idx] = dot_product;
        }
    }
    // CUDA Kernel to find the maximum dot product and its index using atomic operations
    __global__ void find_max_with_index(double *results, double *max_val, int *max_idx, int num_vectors)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        // Initialize shared variables (maximum value and index)
        if (idx < num_vectors)
        {
            double result = results[idx];
            atomicMax_double(max_val, result);

            if (*max_val == result)
            {
                *max_idx = idx; // Update index if the current result is the new max
            }
        }
    }
    // CUDA Kernel to find the maximum dot product, its index, and its key using atomic operations
    __global__ void find_max_with_key(double *results, double *max_val, int *max_idx, int *max_key, const int *key_indices, int num_vectors)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < num_vectors)
        {
            double result = results[idx];
            atomicMax_double(max_val, result);

            // If we find a new maximum, update the index and the key
            if (*max_val == result)
            {
                *max_idx = idx;
                *max_key = key_indices[idx];
            }
        }
    }
    // Host function to find the largest similarity using CUDA
    std::pair<double, int> findLargestSimilarity(const std::vector<std::vector<double>> &vectors, const std::vector<double> &target_vector)
    {
        int num_vectors = vectors.size();
        int vector_size = target_vector.size();

        // Flatten the input vectors into a single array
        std::vector<double> flat_vectors(num_vectors * vector_size);
        for (int i = 0; i < num_vectors; i++)
        {
            for (int j = 0; j < vector_size; j++)
            {
                flat_vectors[i * vector_size + j] = vectors[i][j];
            }
        }

        // Allocate memory on the device (GPU)
        double *d_vectors, *d_target_vector, *d_results, *d_max_val;
        int *d_max_idx;
        hipMalloc(&d_vectors, num_vectors * vector_size * sizeof(double));
        hipMalloc(&d_target_vector, vector_size * sizeof(double));
        hipMalloc(&d_results, num_vectors * sizeof(double));
        hipMalloc(&d_max_val, sizeof(double));
        hipMalloc(&d_max_idx, sizeof(int));

        // Copy data from host to device
        hipMemcpy(d_vectors, flat_vectors.data(), num_vectors * vector_size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_target_vector, target_vector.data(), vector_size * sizeof(double), hipMemcpyHostToDevice);

        // Initialize max_val on the device with a very small value
        double init_max = -DBL_MAX;
        hipMemcpy(d_max_val, &init_max, sizeof(double), hipMemcpyHostToDevice);

        // Set up block and grid sizes
        int block_size = 256;
        int grid_size = (num_vectors + block_size - 1) / block_size;

        // Launch kernel to compute dot products
        compute_dot_products<<<grid_size, block_size>>>(d_vectors, d_target_vector, d_results, num_vectors, vector_size);

        // Launch kernel to find the maximum dot product and its index using atomic operations
        find_max_with_index<<<grid_size, block_size>>>(d_results, d_max_val, d_max_idx, num_vectors);

        // Copy the result back to the host
        double max_similarity;
        int max_idx;
        hipMemcpy(&max_similarity, d_max_val, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&max_idx, d_max_idx, sizeof(int), hipMemcpyDeviceToHost);

        // Free the device memory
        hipFree(d_vectors);
        hipFree(d_target_vector);
        hipFree(d_results);
        hipFree(d_max_val);
        hipFree(d_max_idx);

        // Return the largest similarity and the corresponding index
        return std::make_pair(max_similarity, max_idx);
    }

    std::tuple<double, std::string, int> findSimilaritywithName(
        const std::map<std::string, std::vector<std::vector<double>>> &data_map,
        const std::vector<double> &target_vector)
    {
        int num_vectors = 0;
        int vector_size = target_vector.size();

        // Prepare a flattened version of the vectors and an array of corresponding keys
        std::vector<double> flat_vectors;
        std::vector<int> key_indices;  // Track which map key each vector belongs to
        std::vector<std::string> keys; // Map index to string key
        int key_counter = 0;

        for (const auto &entry : data_map)
        {
            keys.push_back(entry.first); // Keep track of the keys
            for (const auto &vec : entry.second)
            {
                flat_vectors.insert(flat_vectors.end(), vec.begin(), vec.end());
                key_indices.push_back(key_counter);
                num_vectors++;
            }
            key_counter++;
        }

        // Allocate memory on the device (GPU)
        double *d_vectors, *d_target_vector, *d_results, *d_max_val;
        int *d_max_idx, *d_key_indices, *d_max_key;
        hipMalloc(&d_vectors, num_vectors * vector_size * sizeof(double));
        hipMalloc(&d_target_vector, vector_size * sizeof(double));
        hipMalloc(&d_results, num_vectors * sizeof(double));
        hipMalloc(&d_key_indices, num_vectors * sizeof(int));
        hipMalloc(&d_max_val, sizeof(double));
        hipMalloc(&d_max_idx, sizeof(int));
        hipMalloc(&d_max_key, sizeof(int));

        // Copy data from host to device
        hipMemcpy(d_vectors, flat_vectors.data(), num_vectors * vector_size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_target_vector, target_vector.data(), vector_size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_key_indices, key_indices.data(), num_vectors * sizeof(int), hipMemcpyHostToDevice);

        // Initialize max_val on the device with a very small value
        double init_max = -DBL_MAX;
        hipMemcpy(d_max_val, &init_max, sizeof(double), hipMemcpyHostToDevice);

        // Set up block and grid sizes
        int block_size = 256;
        int grid_size = (num_vectors + block_size - 1) / block_size;

        // Launch kernel to compute dot products
        compute_dot_products<<<grid_size, block_size>>>(d_vectors, d_target_vector, d_results, num_vectors, vector_size);

        // Launch kernel to find the maximum dot product, its index, and key
        find_max_with_key<<<grid_size, block_size>>>(d_results, d_max_val, d_max_idx, d_max_key, d_key_indices, num_vectors);

        // Copy the result back to the host
        double max_similarity;
        int max_idx, max_key;
        hipMemcpy(&max_similarity, d_max_val, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&max_idx, d_max_idx, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&max_key, d_max_key, sizeof(int), hipMemcpyDeviceToHost);

        // Free the device memory
        hipFree(d_vectors);
        hipFree(d_target_vector);
        hipFree(d_results);
        hipFree(d_max_val);
        hipFree(d_max_idx);
        hipFree(d_key_indices);
        hipFree(d_max_key);

        // Get the corresponding key and return the result
        std::string max_key_string = keys[max_key];
        return std::make_tuple(max_similarity, max_key_string, max_idx);
    }
}
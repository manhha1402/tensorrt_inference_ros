#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <map>
#include <tuple>
#include <cmath>
#include <string>
#include <limits>
#include <hip/hip_runtime.h>
#include <tensorrt_infer_core/similarity.hpp>

namespace
{
    // CUDA helper functions for cosine similarity calculation
    __device__ double dotProduct(const double *a, const double *b, int size)
    {
        double dot = 0.0;
        for (int i = 0; i < size; ++i)
        {
            dot += a[i] * b[i];
        }
        return dot;
    }

    __device__ double magnitude(const double *vec, int size)
    {
        double sum = 0.0;
        for (int i = 0; i < size; ++i)
        {
            sum += vec[i] * vec[i];
        }
        return sqrt(sum);
    }

    __global__ void computeCosineSimilarity(
        const double *target_vector,
        const double *data,
        int vector_size,
        int total_vectors,
        double *similarities)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < total_vectors)
        {
            const double *current_vector = data + idx * vector_size;
            double dot = dotProduct(target_vector, current_vector, vector_size);
            double mag_target = magnitude(target_vector, vector_size);
            double mag_current = magnitude(current_vector, vector_size);
            similarities[idx] = dot / (mag_target * mag_current + 1e-10); // Avoid division by zero
        }
    }

    // Kernel for finding max similarity (parallel reduction)
    __global__ void findMaxSimilarity(
        const double *similarities,
        int total_vectors,
        double *max_similarity,
        int *max_index)
    {
        extern __shared__ double shared_data[]; // Shared memory for block reduction
        double *shared_similarities = shared_data;
        int *shared_indices = (int *)&shared_data[blockDim.x];

        int tid = threadIdx.x;
        int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

        // Initialize shared memory with similarities and indices
        if (global_idx < total_vectors)
        {
            shared_similarities[tid] = similarities[global_idx];
            shared_indices[tid] = global_idx;
        }
        else
        {
            shared_similarities[tid] = -1e10; // Negative infinity for unused threads
            shared_indices[tid] = -1;
        }
        __syncthreads();

        // Block reduction to find max similarity and its index
        for (int s = blockDim.x / 2; s > 0; s >>= 1)
        {
            if (tid < s && shared_similarities[tid] < shared_similarities[tid + s])
            {
                shared_similarities[tid] = shared_similarities[tid + s];
                shared_indices[tid] = shared_indices[tid + s];
            }
            __syncthreads();
        }

        // Write block's maximum to global memory
        if (tid == 0)
        {
            max_similarity[blockIdx.x] = shared_similarities[0];
            max_index[blockIdx.x] = shared_indices[0];
        }
    }

    // CUDA kernel to compute Euclidean distances
    __global__ void computeEuclideanDistance(const double *d_data, const double *d_target, double *d_distances, size_t vector_size, size_t num_vectors)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < num_vectors)
        {
            double sum = 0.0;
            for (int i = 0; i < vector_size; i++)
            {
                double diff = d_data[idx * vector_size + i] - d_target[i];
                sum += diff * diff;
            }
            d_distances[idx] = sqrt(sum); // Store the Euclidean distance
        }
    }
}

namespace tensorrt_infer_core
{

    // Host function to find the best match using CUDA
    std::tuple<double, std::string, int> findSimilaritywithName(
        const std::map<std::string, std::vector<std::vector<double>>> &data_map,
        const std::vector<double> &target_vector, const std::string &distance_metric)
    {
        hipError_t err;
        std::vector<std::string> keys;
        std::vector<double> flattened_data;
        if (distance_metric == "cosine")
        {

            // Step 1: Flatten the data_map into a contiguous array and track categories and indices
            std::vector<int> category_indices;
            int vector_size = target_vector.size();
            int total_vectors = 0;

            for (const auto &[key, vectors] : data_map)
            {
                keys.push_back(key);
                for (const auto &vec : vectors)
                {
                    flattened_data.insert(flattened_data.end(), vec.begin(), vec.end());
                    category_indices.push_back(keys.size() - 1);
                }
                total_vectors += vectors.size();
            }

            // Step 2: Allocate device memory
            double *d_target_vector, *d_data, *d_similarities;
            double *d_max_similarity; // For storing max similarity
            int *d_max_index;         // For storing max index

            hipMalloc(&d_target_vector, vector_size * sizeof(double));
            hipMalloc(&d_data, flattened_data.size() * sizeof(double));
            hipMalloc(&d_similarities, total_vectors * sizeof(double));
            hipMalloc(&d_max_similarity, sizeof(double) * total_vectors); // Temp max per block
            hipMalloc(&d_max_index, sizeof(int) * total_vectors);         // Temp max indices

            // Step 3: Copy data to device
            hipMemcpy(d_target_vector, target_vector.data(), vector_size * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_data, flattened_data.data(), flattened_data.size() * sizeof(double), hipMemcpyHostToDevice);

            // Step 4: Launch kernel to compute similarities
            int threadsPerBlock = 256;
            int blocksPerGrid = (total_vectors + threadsPerBlock - 1) / threadsPerBlock;
            computeCosineSimilarity<<<blocksPerGrid, threadsPerBlock>>>(
                d_target_vector, d_data, vector_size, total_vectors, d_similarities);
            // Wait for GPU to finish
            hipDeviceSynchronize();
            // Check for kernel errors
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
                return {0.0, "", -1};
            }

            // Step 5: Launch reduction kernel to find the maximum similarity
            size_t shared_memory_size = 2 * threadsPerBlock * sizeof(double); // For similarities and indices
            findMaxSimilarity<<<blocksPerGrid, threadsPerBlock, shared_memory_size>>>(
                d_similarities, total_vectors, d_max_similarity, d_max_index);
            // Wait for GPU to finish
            hipDeviceSynchronize();
            // Check for kernel errors
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
                return {0.0, "", -1};
            }
            // Step 6: Copy block results to host and find global max
            std::vector<double> block_max_similarity(blocksPerGrid);
            std::vector<int> block_max_index(blocksPerGrid);
            hipMemcpy(block_max_similarity.data(), d_max_similarity, blocksPerGrid * sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(block_max_index.data(), d_max_index, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);

            // Step 7: Find the global max similarity on the host
            double best_similarity = -1e10;
            int best_index = -1;
            for (int i = 0; i < blocksPerGrid; ++i)
            {
                if (block_max_similarity[i] > best_similarity)
                {
                    best_similarity = block_max_similarity[i];
                    best_index = block_max_index[i];
                }
            }

            // Step 8: Map best index back to category and local index
            int category_index = category_indices[best_index];
            std::string best_name = keys[category_index];
            int local_index = best_index;
            for (int i = 0; i < category_index; ++i)
            {
                local_index -= data_map.at(keys[i]).size();
            }

            // Step 9: Free device memory
            hipFree(d_target_vector);
            hipFree(d_data);
            hipFree(d_similarities);
            hipFree(d_max_similarity);
            hipFree(d_max_index);

            // Step 10: Return the best match result
            return {best_similarity, best_name, local_index};
        }
        else
        {
            // Flatten the data_map into a 2D vector and collect the keys for each vector
            size_t vector_size = target_vector.size();

            for (const auto &pair : data_map)
            {
                const std::string &key = pair.first;
                for (const auto &vec : pair.second)
                {
                    keys.push_back(key); // Repeat the key for each vector under that key
                    flattened_data.insert(flattened_data.end(), vec.begin(), vec.end());
                }
            }

            size_t num_vectors = flattened_data.size() / vector_size;

            // Allocate device memory for data, target vector, and distances
            double *d_data, *d_target, *d_distances;
            hipMalloc(&d_data, flattened_data.size() * sizeof(double));
            hipMalloc(&d_target, vector_size * sizeof(double));
            hipMalloc(&d_distances, num_vectors * sizeof(double));

            // Copy data and target vector to device
            hipMemcpy(d_data, flattened_data.data(), flattened_data.size() * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_target, target_vector.data(), vector_size * sizeof(double), hipMemcpyHostToDevice);

            // Launch kernel to compute distances
            int blockSize = 256;
            int numBlocks = (num_vectors + blockSize - 1) / blockSize;
            computeEuclideanDistance<<<numBlocks, blockSize>>>(d_data, d_target, d_distances, vector_size, num_vectors);
            // Wait for GPU to finish
            hipDeviceSynchronize();
            // Check for kernel errors
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
                return {0.0, "", -1};
            }
            // Copy distances back to host
            std::vector<double> distances(num_vectors);
            hipMemcpy(distances.data(), d_distances, num_vectors * sizeof(double), hipMemcpyDeviceToHost);

            // Find the index of the smallest distance
            int min_index = std::distance(distances.begin(), std::min_element(distances.begin(), distances.end()));
            double min_distance = distances[min_index];

            // Get the corresponding key for the closest vector
            std::string best_match_key = keys[min_index];

            // Clean up memory
            hipFree(d_data);
            hipFree(d_target);
            hipFree(d_distances);

            // Return the result as a tuple (distance, key, index)
            return std::make_tuple(min_distance, best_match_key, min_index);
        }
    }
}
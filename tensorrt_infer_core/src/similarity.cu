#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cfloat>
// Custom atomicMax for doubles (since CUDA does not support atomicMax for doubles by default)
__device__ double atomicMax_double(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(max(val, __longlong_as_double(assumed))));
    } while (assumed != old);

    return __longlong_as_double(old);
}
__global__ void compute_dot_products(const double* vectors, const double* target_vector, double* dot_products, int num_vectors, int vector_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_vectors) {
        double dot_product = 0.0;
        for (int i = 0; i < vector_size; ++i) {
            dot_product += vectors[idx * vector_size + i] * target_vector[i];
        }
        dot_products[idx] = dot_product;
    }
}
// CUDA Kernel to find the maximum dot product and its index using atomic operations
__global__ void find_max_with_index(double* results, double* max_val, int* max_idx, int num_vectors) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize shared variables (maximum value and index)
    if (idx < num_vectors) {
        double result = results[idx];
        atomicMax_double(max_val, result);

        if (*max_val == result) {
            *max_idx = idx;  // Update index if the current result is the new max
        }
    }
}


// Host function to find the largest similarity using CUDA
std::pair<double, int> findLargestSimilarity(const std::vector<std::vector<double>>& vectors, const std::vector<double>& target_vector) {
   int num_vectors = vectors.size();
    int vector_size = target_vector.size();

    // Flatten the input vectors into a single array
    std::vector<double> flat_vectors(num_vectors * vector_size);
    for (int i = 0; i < num_vectors; i++) {
        for (int j = 0; j < vector_size; j++) {
            flat_vectors[i * vector_size + j] = vectors[i][j];
        }
    }

    // Allocate memory on the device (GPU)
    double *d_vectors, *d_target_vector, *d_results, *d_max_val;
    int *d_max_idx;
    hipMalloc(&d_vectors, num_vectors * vector_size * sizeof(double));
    hipMalloc(&d_target_vector, vector_size * sizeof(double));
    hipMalloc(&d_results, num_vectors * sizeof(double));
    hipMalloc(&d_max_val, sizeof(double));
    hipMalloc(&d_max_idx, sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_vectors, flat_vectors.data(), num_vectors * vector_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_target_vector, target_vector.data(), vector_size * sizeof(double), hipMemcpyHostToDevice);

    // Initialize max_val on the device with a very small value
    double init_max = -DBL_MAX;
    hipMemcpy(d_max_val, &init_max, sizeof(double), hipMemcpyHostToDevice);

    // Set up block and grid sizes
    int block_size = 256;
    int grid_size = (num_vectors + block_size - 1) / block_size;

    // Launch kernel to compute dot products
    compute_dot_products<<<grid_size, block_size>>>(d_vectors, d_target_vector, d_results, num_vectors, vector_size);

    // Launch kernel to find the maximum dot product and its index using atomic operations
    find_max_with_index<<<grid_size, block_size>>>(d_results, d_max_val, d_max_idx, num_vectors);

    // Copy the result back to the host
    double max_similarity;
    int max_idx;
    hipMemcpy(&max_similarity, d_max_val, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&max_idx, d_max_idx, sizeof(int), hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_vectors);
    hipFree(d_target_vector);
    hipFree(d_results);
    hipFree(d_max_val);
    hipFree(d_max_idx);

    // Return the largest similarity and the corresponding index
    return std::make_pair(max_similarity, max_idx);
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cfloat>
#include <map>
#include "tensorrt_infer_core/similarity.hpp"
namespace tensorrt_infer_core
{
    // Custom atomicMax for doubles (since CUDA does not support atomicMax for doubles by default)
    __device__ double atomicMax_double(double *address, double val)
    {
        unsigned long long int *address_as_ull = (unsigned long long int *)address;
        unsigned long long int old = *address_as_ull, assumed;

        do
        {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed, __double_as_longlong(max(val, __longlong_as_double(assumed))));
        } while (assumed != old);

        return __longlong_as_double(old);
    }
    // Custom atomicMin for double precision
    __device__ double atomicMin_double(double* address, double val) {
        unsigned long long int* address_as_ull = (unsigned long long int*) address;
        unsigned long long int old = *address_as_ull, assumed;

        do {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed, __double_as_longlong(fmin(val, __longlong_as_double(assumed))));
        } while (assumed != old);

        return __longlong_as_double(old);
    }
    __global__ void compute_dot_products(const double *vectors, const double *target_vector, double *dot_products, int num_vectors, int vector_size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < num_vectors)
        {
            double dot_product = 0.0;
            for (int i = 0; i < vector_size; ++i)
            {
                dot_product += vectors[idx * vector_size + i] * target_vector[i];
            }
            dot_products[idx] = dot_product;
        }
    }
    // CUDA Kernel to find the maximum dot product and its index using atomic operations
    __global__ void find_max_with_index(double *results, double *max_val, int *max_idx, int num_vectors)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        // Initialize shared variables (maximum value and index)
        if (idx < num_vectors)
        {
            double result = results[idx];
            atomicMax_double(max_val, result);

            if (*max_val == result)
            {
                *max_idx = idx; // Update index if the current result is the new max
            }
        }
    }
    // CUDA Kernel to find the maximum dot product, its index, and its key using atomic operations
    __global__ void find_max_with_key(double *results, double *max_val, int *max_idx, int *max_key, const int *key_indices, int num_vectors)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < num_vectors)
        {
            double result = results[idx];
            atomicMax_double(max_val, result);

            // If we find a new maximum, update the index and the key
            if (*max_val == result)
            {
                *max_idx = idx;
                *max_key = key_indices[idx];
            }
        }
    }

// Kernel to compute the Euclidean distance and find the minimum
    __global__ void findClosestVector(const double* flattened_vectors, const double* target_vector,
                                    int num_vectors, const int* vector_lengths,
                                    const int* vector_offsets, double* min_distance,
                                    int* min_index, int target_size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < num_vectors) {
            int offset = vector_offsets[idx];
            int length = vector_lengths[idx];

            // Compute Euclidean distance
            double dist = 0.0;
            for (int i = 0; i < target_size && i < length; ++i) {
                double diff = target_vector[i] - flattened_vectors[offset + i];
                dist += diff * diff;
            }
            dist = sqrt(dist);

            // Use atomicMin to find the minimum distance
            atomicMin_double(min_distance, dist);
            
            if (dist == *min_distance) {
                atomicExch(min_index, idx);  // Store the index of the minimum
            }
        }
    }


///////////////////////////////
    std::tuple<double, std::string, int> findSimilaritywithName(
        const std::map<std::string, std::vector<std::vector<double>>> &data_map,
        const std::vector<double> &target_vector, const std::string &distance_metric)
    {

        if(distance_metric == "cosine")
        {
        int num_vectors = 0;
        int vector_size = target_vector.size();

        // Prepare a flattened version of the vectors and an array of corresponding keys
        std::vector<double> flat_vectors;
        std::vector<int> key_indices;  // Track which map key each vector belongs to
        std::vector<std::string> keys; // Map index to string key
        int key_counter = 0;

        for (const auto &entry : data_map)
        {
            keys.push_back(entry.first); // Keep track of the keys
            for (const auto &vec : entry.second)
            {
                flat_vectors.insert(flat_vectors.end(), vec.begin(), vec.end());
                key_indices.push_back(key_counter);
                num_vectors++;
            }
            key_counter++;
        }

        // Allocate memory on the device (GPU)
        double *d_vectors, *d_target_vector, *d_results, *d_max_val;
        int *d_max_idx, *d_key_indices, *d_max_key;
        hipMalloc(&d_vectors, num_vectors * vector_size * sizeof(double));
        hipMalloc(&d_target_vector, vector_size * sizeof(double));
        hipMalloc(&d_results, num_vectors * sizeof(double));
        hipMalloc(&d_key_indices, num_vectors * sizeof(int));
        hipMalloc(&d_max_val, sizeof(double));
        hipMalloc(&d_max_idx, sizeof(int));
        hipMalloc(&d_max_key, sizeof(int));

        // Copy data from host to device
        hipMemcpy(d_vectors, flat_vectors.data(), num_vectors * vector_size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_target_vector, target_vector.data(), vector_size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_key_indices, key_indices.data(), num_vectors * sizeof(int), hipMemcpyHostToDevice);

        // Initialize max_val on the device with a very small value
        double init_max = -DBL_MAX;
        hipMemcpy(d_max_val, &init_max, sizeof(double), hipMemcpyHostToDevice);

        // Set up block and grid sizes
        int block_size = 256;
        int grid_size = (num_vectors + block_size - 1) / block_size;

        // Launch kernel to compute dot products
        compute_dot_products<<<grid_size, block_size>>>(d_vectors, d_target_vector, d_results, num_vectors, vector_size);

        // Launch kernel to find the maximum dot product, its index, and key
        find_max_with_key<<<grid_size, block_size>>>(d_results, d_max_val, d_max_idx, d_max_key, d_key_indices, num_vectors);

        // Copy the result back to the host
        double max_similarity;
        int max_idx, max_key;
        hipMemcpy(&max_similarity, d_max_val, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&max_idx, d_max_idx, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&max_key, d_max_key, sizeof(int), hipMemcpyDeviceToHost);

        // Free the device memory
        hipFree(d_vectors);
        hipFree(d_target_vector);
        hipFree(d_results);
        hipFree(d_max_val);
        hipFree(d_max_idx);
        hipFree(d_key_indices);
        hipFree(d_max_key);

        // Get the corresponding key and return the result
        std::string max_key_string = keys[max_key];
        return std::make_tuple(max_similarity, max_key_string, max_idx);
    }
        else if(distance_metric == "euclidean") {
            // Structure to hold the flattened map data on host
            struct FlattenedMap {
                std::vector<double> flattened_vectors;
                std::vector<int> vector_lengths;  // Length of each vector
                std::vector<int> vector_offsets;  // Offsets in the flattened array
                std::vector<std::string> keys;    // Corresponding keys
            };

            // Function to flatten the std::map
            //FlattenedMap flattenMap(const std::map<std::string, std::vector<std::vector<double>>>& my_map) {
                FlattenedMap flattened_data;
                int offset = 0;

                for (const auto& pair : data_map) {
                    flattened_data.keys.push_back(pair.first);
                    for (const auto& vec : pair.second) {
                        flattened_data.flattened_vectors.insert(flattened_data.flattened_vectors.end(), vec.begin(), vec.end());
                        flattened_data.vector_lengths.push_back(vec.size());
                        flattened_data.vector_offsets.push_back(offset);
                        offset += vec.size();
                    }
                }
        
            double* d_flattened_vectors;
            hipMalloc(&d_flattened_vectors, flattened_data.flattened_vectors.size() * sizeof(double));
            hipMemcpy(d_flattened_vectors, flattened_data.flattened_vectors.data(),
                    flattened_data.flattened_vectors.size() * sizeof(double), hipMemcpyHostToDevice);

            double* d_target_vector;
            hipMalloc(&d_target_vector, target_vector.size() * sizeof(double));
            hipMemcpy(d_target_vector, target_vector.data(), target_vector.size() * sizeof(double), hipMemcpyHostToDevice);

            int* d_vector_lengths;
            hipMalloc(&d_vector_lengths, flattened_data.vector_lengths.size() * sizeof(int));
            hipMemcpy(d_vector_lengths, flattened_data.vector_lengths.data(), flattened_data.vector_lengths.size() * sizeof(int), hipMemcpyHostToDevice);

            int* d_vector_offsets;
            hipMalloc(&d_vector_offsets, flattened_data.vector_offsets.size() * sizeof(int));
            hipMemcpy(d_vector_offsets, flattened_data.vector_offsets.data(), flattened_data.vector_offsets.size() * sizeof(int), hipMemcpyHostToDevice);

            // Allocate memory for results on the device
            double* d_min_distance;
            double host_min_distance = DBL_MAX;
            hipMalloc(&d_min_distance, sizeof(double));
            hipMemcpy(d_min_distance, &host_min_distance, sizeof(double), hipMemcpyHostToDevice);

            int* d_min_index;
            hipMalloc(&d_min_index, sizeof(int));
            int host_min_index = -1;
            hipMemcpy(d_min_index, &host_min_index, sizeof(int), hipMemcpyHostToDevice);

            // Launch the kernel
            int num_vectors = flattened_data.vector_offsets.size();
            int threads_per_block = 256;
            int blocks = (num_vectors + threads_per_block - 1) / threads_per_block;
            
            findClosestVector<<<blocks, threads_per_block>>>(d_flattened_vectors, d_target_vector,
                num_vectors, d_vector_lengths, d_vector_offsets, d_min_distance, d_min_index, target_vector.size());

            // Copy results back to host
            double min_distance;
            int min_index;
            hipMemcpy(&min_distance, d_min_distance, sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(&min_index, d_min_index, sizeof(int), hipMemcpyDeviceToHost);

            // Free device memory
            hipFree(d_flattened_vectors);
            hipFree(d_target_vector);
            hipFree(d_vector_lengths);
            hipFree(d_vector_offsets);
            hipFree(d_min_distance);
            hipFree(d_min_index);

            // Get the key associated with the minimum index
            std::string best_key = flattened_data.keys[min_index];

            return std::make_tuple(min_distance, best_key, min_index);
        }
    }
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cfloat>
#include <map>
#include "tensorrt_infer_core/similarity.hpp"
namespace tensorrt_infer_core
{
    // Custom atomicMax for doubles (since CUDA does not support atomicMax for doubles by default)
    __device__ double atomicMax_double(double *address, double val)
    {
        unsigned long long int *address_as_ull = (unsigned long long int *)address;
        unsigned long long int old = *address_as_ull, assumed;

        do
        {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed, __double_as_longlong(max(val, __longlong_as_double(assumed))));
        } while (assumed != old);

        return __longlong_as_double(old);
    }
    // Custom atomicMin for double precision
    __device__ double atomicMin_double(double *address, double val)
    {
        unsigned long long int *address_as_ull = (unsigned long long int *)address;
        unsigned long long int old = *address_as_ull, assumed;

        do
        {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed, __double_as_longlong(fmin(val, __longlong_as_double(assumed))));
        } while (assumed != old);

        return __longlong_as_double(old);
    }
    __global__ void compute_dot_products(const double *vectors, const double *target_vector, double *dot_products, int num_vectors, int vector_size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < num_vectors)
        {
            double dot_product = 0.0;
            for (int i = 0; i < vector_size; ++i)
            {
                dot_product += vectors[idx * vector_size + i] * target_vector[i];
            }
            dot_products[idx] = dot_product;
        }
    }
    // CUDA Kernel to find the maximum dot product and its index using atomic operations
    __global__ void find_max_with_index(double *results, double *max_val, int *max_idx, int num_vectors)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        // Initialize shared variables (maximum value and index)
        if (idx < num_vectors)
        {
            double result = results[idx];
            atomicMax_double(max_val, result);

            if (*max_val == result)
            {
                *max_idx = idx; // Update index if the current result is the new max
            }
        }
    }
    // CUDA Kernel to find the maximum dot product, its index, and its key using atomic operations
    __global__ void find_max_with_key(double *results, double *max_val, int *max_idx, int *max_key, const int *key_indices, int num_vectors)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < num_vectors)
        {
            double result = results[idx];
            atomicMax_double(max_val, result);

            // If we find a new maximum, update the index and the key
            if (*max_val == result)
            {
                *max_idx = idx;
                *max_key = key_indices[idx];
            }
        }
    }

    // Kernel to compute the Euclidean distance and find the minimum

    // CUDA kernel to compute Euclidean distances
    __global__ void computeEuclideanDistance(const double *d_data, const double *d_target, double *d_distances, size_t vector_size, size_t num_vectors)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < num_vectors)
        {
            double sum = 0.0;
            for (int i = 0; i < vector_size; i++)
            {
                double diff = d_data[idx * vector_size + i] - d_target[i];
                sum += diff * diff;
            }
            d_distances[idx] = sqrt(sum); // Store the Euclidean distance
        }
    }

    ///////////////////////////////
    std::tuple<double, std::string, int>
    findSimilaritywithName(
        const std::map<std::string, std::vector<std::vector<double>>> &data_map,
        const std::vector<double> &target_vector, const std::string &distance_metric)
    {

        if (distance_metric == "cosine")
        {
            int num_vectors = 0;
            int vector_size = target_vector.size();

            // Prepare a flattened version of the vectors and an array of corresponding keys
            std::vector<double> flat_vectors;
            std::vector<int> key_indices;  // Track which map key each vector belongs to
            std::vector<std::string> keys; // Map index to string key
            int key_counter = 0;

            for (const auto &entry : data_map)
            {
                keys.push_back(entry.first); // Keep track of the keys
                for (const auto &vec : entry.second)
                {
                    flat_vectors.insert(flat_vectors.end(), vec.begin(), vec.end());
                    key_indices.push_back(key_counter);
                    num_vectors++;
                }
                key_counter++;
            }

            // Allocate memory on the device (GPU)
            double *d_vectors, *d_target_vector, *d_results, *d_max_val;
            int *d_max_idx, *d_key_indices, *d_max_key;
            hipMalloc(&d_vectors, num_vectors * vector_size * sizeof(double));
            hipMalloc(&d_target_vector, vector_size * sizeof(double));
            hipMalloc(&d_results, num_vectors * sizeof(double));
            hipMalloc(&d_key_indices, num_vectors * sizeof(int));
            hipMalloc(&d_max_val, sizeof(double));
            hipMalloc(&d_max_idx, sizeof(int));
            hipMalloc(&d_max_key, sizeof(int));

            // Copy data from host to device
            hipMemcpy(d_vectors, flat_vectors.data(), num_vectors * vector_size * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_target_vector, target_vector.data(), vector_size * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_key_indices, key_indices.data(), num_vectors * sizeof(int), hipMemcpyHostToDevice);

            // Initialize max_val on the device with a very small value
            double init_max = -DBL_MAX;
            hipMemcpy(d_max_val, &init_max, sizeof(double), hipMemcpyHostToDevice);

            // Set up block and grid sizes
            int block_size = 256;
            int grid_size = (num_vectors + block_size - 1) / block_size;

            // Launch kernel to compute dot products
            compute_dot_products<<<grid_size, block_size>>>(d_vectors, d_target_vector, d_results, num_vectors, vector_size);

            // Launch kernel to find the maximum dot product, its index, and key
            find_max_with_key<<<grid_size, block_size>>>(d_results, d_max_val, d_max_idx, d_max_key, d_key_indices, num_vectors);

            // Copy the result back to the host
            double max_similarity;
            int max_idx, max_key;
            hipMemcpy(&max_similarity, d_max_val, sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(&max_idx, d_max_idx, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&max_key, d_max_key, sizeof(int), hipMemcpyDeviceToHost);

            // Free the device memory
            hipFree(d_vectors);
            hipFree(d_target_vector);
            hipFree(d_results);
            hipFree(d_max_val);
            hipFree(d_max_idx);
            hipFree(d_key_indices);
            hipFree(d_max_key);

            // Get the corresponding key and return the result
            std::string max_key_string = keys[max_key];
            return std::make_tuple(max_similarity, max_key_string, max_idx);
        }
        else if (distance_metric == "euclidean")
        {
            // Flatten the data_map into a 2D vector and collect the keys for each vector
            std::vector<std::string> keys;
            std::vector<double> flattened_data;
            size_t vector_size = target_vector.size();

            for (const auto &pair : data_map)
            {
                const std::string &key = pair.first;
                for (const auto &vec : pair.second)
                {
                    keys.push_back(key); // Repeat the key for each vector under that key
                    flattened_data.insert(flattened_data.end(), vec.begin(), vec.end());
                }
            }

            size_t num_vectors = flattened_data.size() / vector_size;

            // Allocate device memory for data, target vector, and distances
            double *d_data, *d_target, *d_distances;
            hipMalloc(&d_data, flattened_data.size() * sizeof(double));
            hipMalloc(&d_target, vector_size * sizeof(double));
            hipMalloc(&d_distances, num_vectors * sizeof(double));

            // Copy data and target vector to device
            hipMemcpy(d_data, flattened_data.data(), flattened_data.size() * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_target, target_vector.data(), vector_size * sizeof(double), hipMemcpyHostToDevice);

            // Launch kernel to compute distances
            int blockSize = 256;
            int numBlocks = (num_vectors + blockSize - 1) / blockSize;
            computeEuclideanDistance<<<numBlocks, blockSize>>>(d_data, d_target, d_distances, vector_size, num_vectors);

            // Copy distances back to host
            std::vector<double> distances(num_vectors);
            hipMemcpy(distances.data(), d_distances, num_vectors * sizeof(double), hipMemcpyDeviceToHost);

            // Find the index of the smallest distance
            int min_index = std::distance(distances.begin(), std::min_element(distances.begin(), distances.end()));
            double min_distance = distances[min_index];

            // Get the corresponding key for the closest vector
            std::string best_match_key = keys[min_index];

            // Clean up memory
            hipFree(d_data);
            hipFree(d_target);
            hipFree(d_distances);

            // Return the result as a tuple (distance, key, index)
            return std::make_tuple(min_distance, best_match_key, min_index);
        }
    }
}
